#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

/* Bounds of the Mandelbrot set */
#define X_MIN -1.78
#define X_MAX 0.78
#define Y_MIN -0.96
#define Y_MAX 0.96

typedef struct {
    int nb_rows, nb_columns; /* Dimensions */
    char * pixels; /* Linearized matrix of pixels */
} Image;

void error_options () {
    fprintf (stderr, "Use : ./mandel [options]\n\n");
    fprintf (stderr, "Options \t Meaning \t\t Default val.\n\n");
    fprintf (stderr, "-n \t\t Nb iter. \t\t 100\n");
    fprintf (stderr, "-b \t\t Bounds \t\t -1.78 0.78 -0.96 0.96\n");
    fprintf (stderr, "-d \t\t Dimensions \t\t 1024 768\n");
    fprintf (stderr, "-f \t\t File \t\t /tmp/mandel.ppm\n");
    exit (1);
}

void analyzis(int argc, char **argv, int *nb_iter, double *x_min, double *x_max, double *y_min, double *y_max, int *width, int *height, const char **path) {
    const char *opt = "b:d:n:f:";
    int c;

    /* Default values */
    *nb_iter = 200000;
    *x_min = X_MIN;
    *x_max = X_MAX;
    *y_min = Y_MIN;
    *y_max = Y_MAX;
    *width = 1024;
    *height = 768;
    *path = "mandel_cuda.ppm";  // Assign directly to a const char*

    /* Analysis of arguments */
    while ((c = getopt(argc, argv, opt)) != EOF) {
        switch (c) {
            case 'b':
                sscanf(optarg, "%lf", x_min);
                sscanf(argv[optind++], "%lf", x_max);
                sscanf(argv[optind++], "%lf", y_min);
                sscanf(argv[optind++], "%lf", y_max);
                break;
            case 'd': /* width */
                sscanf(optarg, "%d", width);
                sscanf(argv[optind++], "%d", height);
                break;
            case 'n': /* Number of iterations */
                *nb_iter = atoi(optarg);
                break;
            case 'f': /* Output file */
                *path = optarg;
                break;
            default:
                error_options();
        }
    }
}

void initialization (Image * im, int nb_columns, int nb_rows) {
    im -> nb_rows = nb_rows;
    im -> nb_columns = nb_columns;
    im -> pixels = (char *) malloc (sizeof (char) * nb_rows * nb_columns); /* Space memory allocation */
}

void save (const Image * im, const char * path) {
    /* Image saving using the ASCII format'.PPM' */
    unsigned i;
    FILE * f = fopen (path, "w");
    fprintf (f, "P6\n%d %d\n255\n", im -> nb_columns, im -> nb_rows);
    for (i = 0; i < im -> nb_columns * im -> nb_rows; i ++) {
        char c = im -> pixels [i];
        fprintf (f, "%c%c%c", c, c, c); /* Monochrome weight */
    }
    fclose (f);
}


//Cuda function
__global__ void Compute(char *pixels, int nb_columns, int nb_rows, int nb_iter, double x_min, double x_max, double y_min, double y_max){
    int pos = blockIdx.x * blockDim.x + threadIdx.x; //index of a thread in our grid 

    if (pos >= nb_rows * nb_columns) return; // check that we are in the grid

    //INdex of our lines and columns
    int l = pos/nb_columns; 
    int c = pos%nb_columns;  

    //In Cuda we don't need loop anymore, every thread is associated to the computation of a pixel
    /* Discretization */
    double dx = (x_max - x_min) / nb_columns;
    double dy = (y_max - y_min) / nb_rows;

    /* Computation at each point of the image */
    double a = x_min + c * dx;
    double b = y_max - l * dy;
    double x = 0, y = 0;
    int i = 0;

    while (i < nb_iter) {
        double tmp = x;
        x = x * x - y * y + a;
        y = 2 * tmp * y + b;
        if (x * x + y * y > 4){ /* Divergence ! */
            break;
        } else {
            i++;
        }
    }
    pixels[pos] = (char)((i * 255)/nb_iter);
}

int main(int argc, char **argv) {
    int nb_iter, width, height;
    double x_min, x_max, y_min, y_max;
    const char *path;  // Declare as const char*
    Image im;

    struct timespec tstart, tend, tstart_copy, tend_copy, tstart_kernel, tend_kernel;
    clock_gettime(CLOCK_MONOTONIC, &tstart);

    analyzis(argc, argv, &nb_iter, &x_min, &x_max, &y_min, &y_max, &width, &height, &path);

    initialization(&im, width, height);

    char *d_pixels;
    size_t size = width * height * sizeof(char); //total size of the image we have with*height pixels and each of them of size(char)

    //------Time for memory allocation and copying to the GPU------
    clock_gettime(CLOCK_MONOTONIC, &tstart_copy);

    hipMalloc((void **)&d_pixels, size);     //Allocation of memory into the GPU to sotre pixels
    hipMemcpy(d_pixels, im.pixels, size, hipMemcpyHostToDevice); //Copy of the memory into the GPU

    clock_gettime(CLOCK_MONOTONIC, &tend_copy);

    //-------Measure time taken for memory copy-------
    double elap_copy_time = (tend_copy.tv_sec - tstart_copy.tv_sec) + (tend_copy.tv_nsec - tstart_copy.tv_nsec)/1e9f;
    printf("Time taken for memory copy to GPU: %2.9lf seconds\n", elap_copy_time);

    //Define block and grid
    int total_pixels = width * height; //total pixels of the image
    int threads_per_block = 256; // we choose 256 threads per block (arbitrary)
    int num_blocks = (total_pixels + threads_per_block - 1)/threads_per_block; //total block needed to cover all the pixels

    //Measure time taken for GPU computation
    //------------------------------------------------
    clock_gettime(CLOCK_MONOTONIC, &tstart_kernel);

    //Perform CUDA computation
    Compute<<<num_blocks, threads_per_block>>>(d_pixels, width, height, nb_iter, x_min, x_max, y_min, y_max);

    clock_gettime(CLOCK_MONOTONIC, &tend_kernel); 
    //-------------------------------------------------

    //------Measure time taken for GPU computation------
    double elap_kernel_time = (tend_kernel.tv_sec - tstart_kernel.tv_sec) + (tend_kernel.tv_nsec - tstart_kernel.tv_nsec)/1e9f;
    printf("Time taken for GPU computation: %2.9lf seconds\n", elap_kernel_time);

    //Copy the result to the CPU
    hipMemcpy(im.pixels, d_pixels, size, hipMemcpyDeviceToHost);

    //Save the final image
    save(&im, path);

    // Free memory on the GPU
    hipFree(d_pixels);

    //--------Measure total time taken--------
    clock_gettime(CLOCK_MONOTONIC, &tend);
    double elap_time = (tend.tv_sec - tstart.tv_sec) + (tend.tv_nsec - tstart.tv_nsec)/1e9f;
    printf("Total elapsed time: %2.9lf seconds\n", elap_time);

    return 0;
}
